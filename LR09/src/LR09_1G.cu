#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

int main() {
    int num = 1 << 12;
    int size = 32 * num;
    float *device, *hostPinned, *host, time = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    host = (float *) malloc(size * sizeof(float));
    hipHostMalloc((void **) &hostPinned, size * sizeof(float), hipHostMallocDefault);
    hipMalloc((void **) &device, size * sizeof(float));
    hipMemset(device, 1024, size * sizeof(float));
    hipMemcpy(device, host, size * sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(start, nullptr);
    hipMemcpy(host, device, size * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    cout << "Стандартное копирование с device на host:             "<< time << endl;

    hipStream_t stream;
    hipStreamCreate(&stream);
    hipEventRecord(start, nullptr);
    hipMemcpyAsync(hostPinned, device, size * sizeof(float), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
    time = 0;
    hipEventElapsedTime(&time, start, stop);
    cout << "Закрепленная память (pinned memory) с device на хост: "<< time  << endl;

    hipEventRecord(start, nullptr);
    hipMemcpy(device, host, size * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
    time = 0;
    hipEventElapsedTime(&time, start, stop);
    cout << "Стандартное копирование с host на device:             " << time << endl;

    hipEventRecord(start, nullptr);
    hipMemcpyAsync(device, hostPinned, size * sizeof(float), hipMemcpyHostToDevice, stream);
    hipStreamSynchronize(stream);
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
    time = 0;
    hipEventElapsedTime(&time, start, stop);
    cout << "Закрепленная память (pinned memory) с хост на device: " << time << endl;

    hipFree(device);
    hipHostFree(host);
    free(host);
    return 0;
}

