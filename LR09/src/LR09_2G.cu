#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

void show_mass(float *a, int num){
    for (int i = 0; i < num; i++) {
        printf("%f ",a[i]);
        if (i%10 == 0) printf("\n");
    }
    printf("\n");
}
__global__ void addVectors(float *a, float *b, float *c, int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        c[i] = a[i] + b[i];
    }
}

int main() {
    int stream_num = 1;
    int num = 1 << 12;
    int size = 32 * num;
    int portion_size = size / stream_num;

    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    float time = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    auto *streams = (hipStream_t*)calloc(stream_num, sizeof(hipStream_t));
    for (int i = 0; i < stream_num; i++) hipStreamCreate(&streams[i]);

    hipHostMalloc((void **) &h_a, size * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **) &h_b, size * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void **) &h_c, size * sizeof(float), hipHostMallocDefault);
    hipMalloc((void **)&d_a, size * sizeof(float));
    hipMalloc((void **)&d_b, size * sizeof(float));
    hipMalloc((void **)&d_c, size * sizeof(float));

    for (int i = 0; i < size; i++) {
        h_a[i] = i;
        h_b[i] = i + 1;
    }

    for (int i = 0; i < stream_num; i++) {
        hipMemcpyAsync(d_a + i * portion_size, h_a + i * portion_size, portion_size * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_b + i * portion_size, h_b + i * portion_size, portion_size * sizeof(float), hipMemcpyHostToDevice, streams[i]);
    }
    for (int i = 0; i < stream_num; i++) hipStreamSynchronize(streams[i]);

    dim3 threadsPerBlock(256);
    dim3 numBlocks((portion_size + threadsPerBlock.x - 1) / threadsPerBlock.x);

    hipEventRecord(start, nullptr);
    for (int i = 0; i < stream_num; i++)
        addVectors<<<numBlocks, threadsPerBlock, 0, streams[i]>>>(d_a + i * portion_size, d_b + i * portion_size, d_c + i * portion_size, portion_size);

    for (int i = 0; i < stream_num; i++) {
        hipMemcpyAsync(h_c + i * portion_size, d_c + i * portion_size, portion_size * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
    }
    for (int i = 0; i < stream_num; i++) hipStreamSynchronize(streams[i]);

    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    cout << "time = "<< time << endl;

    //show_mass(h_c, 100);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(h_a);
    hipFree(h_b);
    hipFree(h_c);
    for (int i = 0; i < stream_num; i++) hipStreamDestroy(streams[i]);

    return 0;
}
