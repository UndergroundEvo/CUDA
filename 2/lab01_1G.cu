//
// Created by miron on 06.02.24.
//

#include <iostream>
#include <hip/hip_runtime.h>
const long long N = 999989;

__global__ void vectorAdd(const float *a, const float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;

    hipMalloc((void **)&d_a, N * sizeof(float));
    hipMalloc((void **)&d_b, N * sizeof(float));
    hipMalloc((void **)&d_c, N * sizeof(float));

    a = (float *)malloc(N * sizeof(float));
    b = (float *)malloc(N * sizeof(float));
    c = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Копирование данных с хоста на устройство
    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    // Выполнение ядра CUDA для сложения векторов
    vectorAdd<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);

    // Копирование результата с устройства на хост
    hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Вывод результата
    for (int i = 0; i < 10; ++i) {
        std::cout << "c[" << i << "] = " << c[i] << std::endl;
    }

    // Освобождение памяти
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}




