#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

__global__ void vectorAdd(const float* a, const float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 10000;
    float elapsedTime;
    hipEvent_t start, stop;

    float* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));
    hipMalloc((void**)&d_c, n * sizeof(float));

    float* h_a = new float[n];
    float* h_b = new float[n];
    for (int i = 0; i < n; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);


    // Вычисляем количество блоков и нитей на блок
    int blockSize = 8192;
    int numBlocks = n;

    //int numBlocks = (n + blockSize - 1) / blockSize;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    vectorAdd << < numBlocks, blockSize >> > (d_a, d_b, d_c, n);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    float* h_c = new float[n];
    hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    // Выводим результат
//    std::cout << "Result: ";
//    for (int i = 0; i < n; ++i) {
//        std::cout << h_c[i] << " ";
//    }
//    std::cout << std::endl;
    std::cout << elapsedTime << std::endl;


    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
