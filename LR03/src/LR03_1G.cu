#include <iostream>
#include <stdio.h>
#include <chrono>

#include "hip/hip_runtime.h"


using namespace std;

const int n = 1 << 20;

typedef std::chrono::milliseconds ms;
typedef std::chrono::nanoseconds ns;

__global__ void vectorAdd(const float *a, const float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

int main() {
    float elapsedTime;
    int blockSize = 1024;
    hipEvent_t start, stop;
    chrono::time_point<chrono::system_clock> start_chrono, end_chrono;

    int numBlocks;
    cout << "Enter threads num: ";
    cin >> numBlocks;

    float *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, n * sizeof(float));
    hipMalloc((void **) &d_b, n * sizeof(float));
    hipMalloc((void **) &d_c, n * sizeof(float));

    float *h_a = new float[n],
            *h_b = new float[n];
    for (int i = 0; i < n; ++i) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    start_chrono = chrono::system_clock::now();
    vectorAdd<<<dim3(numBlocks), blockSize>>>(d_a, d_b, d_c, n);
    hipEventRecord(stop, 0);
    end_chrono = chrono::system_clock::now();

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    float *h_c = new float[n];
    hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    cout << "CUDA Event time:\n\t"
         << elapsedTime
         << endl;

    cout << "Chrono time:\n\t"
         << chrono::duration_cast<ms>(end_chrono - start_chrono).count() << "ms\n\t"
         << chrono::duration_cast<ns>(end_chrono - start_chrono).count() << "ns"
         << endl;

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
