/*
#include <iostream>
#include <iomanip>
#include <cuda_runtime.h>
#include <cublas_v2.h>

__global__ void matrixMultiplication(float *a, float *b, float *c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int i = 0; i < N; ++i) {
            sum += a[row * N + i] * b[i * N + col];
        }
        c[row * N + col] = sum;
    }
}
void initMatrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            matrix[i * cols + j] = i + j;
        }
    }
}
void printMatrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << matrix[i * cols + j] << "\t";
        }
        std::cout << std::endl;
    }
}

int main() {
    const int num =  3;
    int N = 3 * num;
    float elapsedTime = 0;
    cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);

    float *h_A = new float[N * N];
    float *h_B = new float[N * N];
    float *h_C = new float[N * N];

    initMatrix(h_A, N, N);
    initMatrix(h_B, N, N);

    float *d_A, *d_B, *d_C;
    cudaMalloc(&d_A, N * N * sizeof(float));
    cudaMalloc(&d_B, N * N * sizeof(float));
    cudaMalloc(&d_C, N * N * sizeof(float));

    cudaMemcpy(d_A, h_A, N * N * sizeof(float), cudaMemcpyHostToDevice);
    cudaMemcpy(d_B, h_B, N * N * sizeof(float), cudaMemcpyHostToDevice);

    dim3 threadsPerBlock(1024, 1024);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    cudaEventRecord(start, 0);
    matrixMultiplication<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    cudaDeviceSynchronize();
    cudaEventRecord(stop, 0);

    cudaEventSynchronize(stop);
    cudaEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time using CUDA code: " <<  std::setprecision(15) << elapsedTime <<  std::endl;

    cudaMemcpy(h_C, d_C, N * N * sizeof(float), cudaMemcpyDeviceToHost);

    std::cout << "Matrix A:" << std::endl;
    printMatrix(h_A, N, N);
    std::cout << std::endl;

    std::cout << "Matrix B:" << std::endl;
    printMatrix(h_B, N, N);
    std::cout << std::endl;

    std::cout << "End matrix C:" << std::endl;
    printMatrix(h_C, N, N);

    cudaFree(d_A);
    cudaFree(d_B);
    cudaFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    return 0;
}*/

#include <iostream>
#include <hip/hip_runtime.h>
#include <iomanip>

__global__ void matrixMultiply(int *a, int *b, int *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int i = 0; i < n; ++i) {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main() {
    const int num =  1 << 2;
    int N = 3 * num;
    float elapsedTime = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *h_a, *h_b, *h_c;
    h_a = new int[N * N];
    h_b = new int[N * N];
    h_c = new int[N * N];

    for (int i = 0; i < N * N; ++i) {
        h_a[i] = i;
        h_b[i] = i;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, N * N * sizeof(int));
    hipMalloc((void**)&d_b, N * N * sizeof(int));
    hipMalloc((void**)&d_c, N * N * sizeof(int));

    hipMemcpy(d_a, h_a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(1024, 1024);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    hipEventRecord(start, 0);
    matrixMultiply<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);
    //cudaDeviceSynchronize();
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time using CUDA code: " <<  std::setprecision(15) << elapsedTime <<  std::endl;

    // Печать результата
    std::cout << "Result Matrix:" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            std::cout << h_c[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    hipMemcpy(h_c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}