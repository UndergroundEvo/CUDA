#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void initMatrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            matrix[i * cols + j] = i + j;
        }
    }
}
void printMatrix(float *matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << matrix[i * cols + j] << "\t";
        }
        std::cout << std::endl;
    }
}

int main() {
    const int num =  1 << 14;
    int N = 2 * num;
    float elapsedTime = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *h_A = new float[N * N];
    float *h_B = new float[N * N];
    float *h_C = new float[N * N];

    initMatrix(h_A, N, N);
    initMatrix(h_B, N, N);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    float alpha = 1.0f, beta = 0.0f;

    hipEventRecord(start, 0);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Time using cuBLAS code: " /*<<  std::setprecision(15)*/ << elapsedTime <<  std::endl;

    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

/*    std::cout << "Matrix A:" << std::endl;
    printMatrix(h_A, N, N);
    std::cout << std::endl;

    std::cout << "Matrix B:" << std::endl;
    printMatrix(h_B, N, N);
    std::cout << std::endl;

    std::cout << "End matrix C:" << std::endl;
    printMatrix(h_C, N, N);*/

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    return 0;
}
